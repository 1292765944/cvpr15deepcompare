#include "hip/hip_runtime.h"
// Copyright 2015 Sergey Zagoruyko, Nikos Komodakis
// sergey.zagoruyko@imagine.enpc.fr, nikos.komodakis@enpc.fr
// Ecole des Ponts ParisTech, Universite Paris-Est, IMAGINE
//
// The software is free to use only for non-commercial purposes.
// IF YOU WOULD LIKE TO USE IT FOR COMMERCIAL PURPOSES, PLEASE CONTACT
// Prof. Nikos Komodakis (nikos.komodakis@enpc.fr)
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/features2d.hpp>

#include <iostream>
#include <THC/THC.h>
#include <cunn.h>
#include "loader.h"

// defined by network architecture
#define M 64

// Given an image an coordinates+sizes of detected points
// extract corresponding image patches with OpenCV functions
// input image is in [0 255] range,
// the patches are divided by 255 and mean-normalized
// Note: depending on the type of applications you might want to use 
// orientation of detected region or inrease the cropped bounding box by some
// constant
// Another note: this is of course not the fastest way to extracted features.
// The ultimate would be to use CUDA texture memory and process all the features
// from am image in parallel
void extractPatches(const cv::Mat& image,
    const std::vector<cv::KeyPoint>& kp,
    std::vector<cv::Mat>& patches)
{
  for(auto &it : kp)
  {
    cv::Mat patch(M, M, CV_32F);
    cv::Mat buf;
    // increase the size of the region to include some context
    cv::getRectSubPix(image, cv::Size(it.size*1.3, it.size*1.3), it.pt, buf);
    cv::Scalar m = cv::mean(buf);
    cv::resize(buf, patch, cv::Size(M,M));
    patch.convertTo(patch, CV_32F, 1./255.);
    patch = patch.isContinuous() ? patch : patch.clone();
    // mean subtraction is crucial!
    patches.push_back(patch - m[0]/255.);
  }
}

// Copy extracted patches to CUDA memory and run the network
// One has to keep mind that GPU memory is limited and extracting too many patches
// at once might cause troubles
// So if you need to extract a lot of patches, an efficient way would be to
// devide the set in smaller equal parts and preallocate CPU and GPU memory
void extractDescriptors(THCState *state,
    cunn::Sequential::Ptr net,
    const std::vector<cv::Mat>& patches,
    cv::Mat& descriptors)
{
  size_t N = patches.size();
  THFloatTensor *buffer = THFloatTensor_newWithSize4d(N, 1, M, M);
  float *data = THFloatTensor_data(buffer);

  for(size_t i = 0; i < N; ++i)
    memcpy(data + i*M*M, patches[i].data, sizeof(float) * M * M);

  // initialize 4D CUDA tensor and copy patches into it
  THCudaTensor *input = THCudaTensor_newWithSize4d(state, N, 1, M, M);
  THCudaTensor_copyFloat(state, input, buffer);

  // propagate through the network
  THCudaTensor *output = net->forward(input);

  // copy descriptors back
  THFloatTensor *desc = THFloatTensor_newWithSize2d(N, output->size[1]);
  THFloatTensor_copyCuda(state, desc, output);

  descriptors.create(N, output->size[1], CV_32F);
  memcpy(descriptors.data, THFloatTensor_data(desc), sizeof(float) * N * output->size[1]);

  THCudaTensor_free(state, input);
  THFloatTensor_free(buffer);
  THFloatTensor_free(desc);
}


int main(int argc, char** argv)
{
  THCState *state = (THCState*)malloc(sizeof(THCState));
  THCudaInit(state);

  const char *network_path = "/opt/projects/deepfeat/release/networks/siam/siam_desc_notredame.bin";
  auto net = loadNetwork(state, network_path);

  // load the images
  cv::Mat ima = cv::imread("../100_7100.JPG");
  cv::Mat imb = cv::imread("../100_7101.JPG");

  if(ima.empty() || imb.empty())
  {
    std::cout << "images not found\n";
    return 1;
  }

  cv::Mat ima_gray, imb_gray;
  cv::cvtColor(ima, ima_gray, cv::COLOR_BGR2GRAY);
  cv::cvtColor(imb, imb_gray, cv::COLOR_BGR2GRAY);

  // Here we set min_area parameter to a bigger value, like that minimal size
  // of a patch will be around 11x11, because the network was trained on bigger patches
  // this parameter is important in practice
  cv::Ptr<cv::MSER> detector = cv::MSER::create(5, 120);
  std::vector<cv::KeyPoint> kpa, kpb;
  detector->detect(ima_gray, kpa);
  detector->detect(imb_gray, kpb);
  std::cout << "image A MSER points detected: " << kpa.size() << std::endl;
  std::cout << "image B MSER points detected: " << kpb.size() << std::endl;

  std::vector<cv::Mat> patches_a, patches_b;
  extractPatches(ima_gray, kpa, patches_a);
  extractPatches(imb_gray, kpb, patches_b);

  cv::Mat descriptors_a, descriptors_b;
  extractDescriptors(state, net, patches_a, descriptors_a);
  extractDescriptors(state, net, patches_b, descriptors_b);

  cv::FlannBasedMatcher matcher;
  std::vector<cv::DMatch> matches;
  matcher.match( descriptors_a, descriptors_b, matches );

  double max_dist = 0; double min_dist = 100;

  //-- Quick calculation of max and min distances between keypoints
  for( int i = 0; i < descriptors_a.rows; i++ )
  { double dist = matches[i].distance;
    if( dist < min_dist ) min_dist = dist;
    if( dist > max_dist ) max_dist = dist;
  }

  printf("-- Max dist : %f \n", max_dist );
  printf("-- Min dist : %f \n", min_dist );


  std::vector<cv::DMatch> good_matches;
  for( int i = 0; i < descriptors_a.rows; i++ )
  { if( matches[i].distance <= max(4*min_dist, 0.02) )
    { good_matches.push_back( matches[i]); }
  }

  //-- Draw only "good" matches
  cv::Mat img_matches;
  cv::drawMatches( ima, kpa, imb, kpb,
               good_matches, img_matches, cv::Scalar::all(-1), cv::Scalar::all(-1),
               std::vector<char>(), cv::DrawMatchesFlags::NOT_DRAW_SINGLE_POINTS );

  for(auto &it : kpa)
    cv::circle(ima, cv::Point(it.pt.x, it.pt.y), it.size, cv::Scalar(255,255,0));
  for(auto &it : kpb)
    cv::circle(imb, cv::Point(it.pt.x, it.pt.y), it.size, cv::Scalar(255,255,0));

  cv::imshow("matches", img_matches);
  cv::imshow("keypoints image 1", ima);
  cv::imshow("keypoints image 2", imb);
  cv::waitKey();
  THCudaShutdown(state);

  return 0;
}
